#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>
#include <parallel/algorithm>

#include "HybridDBSCAN.cuh"

using std::ceil;
using std::copy;
using std::cout;
using std::endl;
using std::iostream;
using std::lower_bound;
using std::max;
using std::min;
using std::ofstream;
using thrust::device_ptr;
using thrust::device_vector;
using thrust::raw_pointer_cast;
using thrust::sort_by_key;

HybridDBSCAN::HybridDBSCAN(
    const float epsilon,
    const uint minpts,
    DataPointsType dataPoints,
    uint dataSize,
    uint blockSize) : epsilon(epsilon),
                      epsilonPow(pow(epsilon, 2)),
                      minpts(minpts),
                      dataPoints(dataPoints),
                      dataSize(dataSize),
                      blockSize(blockSize),
                      gridSize(0),
                      orderedIDKey(nullptr),
                      orderedIDValue(nullptr),
                      neighborsCnt(0) {
    clusterIDs.resize(dataSize);
    for (int i = 0; i < dataSize; i++) {
        clusterIDs[i] = HybridDBSCAN::UNVISITED;
    }
}

void HybridDBSCAN::calcCells() {
    // minVals, maxVals
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] = dataPoints[dim][0];
        maxVals[dim] = dataPoints[dim][0];
    }
    for (uint dim = 0; dim < 2; dim++) {
        for (uint i = 0; i < dataSize; i++) {
            minVals[dim] = min(minVals[dim], dataPoints[dim][i]);
            maxVals[dim] = max(maxVals[dim], dataPoints[dim][i]);
        }
    }
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] -= epsilon;
        maxVals[dim] += epsilon;
    }
    // nCells
    totalCells = 1;
    for (uint dim = 0; dim < 2; dim++) {
        nCells[dim] = ceil((maxVals[dim] - minVals[dim]) / epsilon);
        totalCells *= nCells[dim];
    }
}

void HybridDBSCAN::constructIndex() {
    calcCells();

    // calculate cellIDs of grid cells
    auto cellIDs_dup = vector<uint64>();
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        cellIDs_dup.push_back(cellID);
    }
    omp_set_num_threads(NUM_THREADS);
    __gnu_parallel::sort(cellIDs_dup.begin(), cellIDs_dup.end());
    auto cellIDs_uqe = vector<uint64>();
    cellIDs_uqe.push_back(cellIDs_dup[0]);
    for (uint i = 1; i < cellIDs_dup.size(); i++) {
        if (cellIDs_dup[i] != cellIDs_dup[i - 1]) {
            cellIDs_uqe.push_back(cellIDs_dup[i]);
        }
    }

    grid2CellID.resize(cellIDs_uqe.size());
    for (uint i = 0; i < cellIDs_uqe.size(); i++) {
        grid2CellID[i] = cellIDs_uqe[i];
    }

    // get dataIDs per non empty cell(aka. grid)
    gridSize = cellIDs_uqe.size();
    auto gridDataIDs = vector<vector<uint64>>(gridSize);  // dataID per cell
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        if (cellID > totalCells) {
            fprintf(stderr, "ERROR: LinearID = %lu > totalCells = %lu\n", cellID, totalCells);
        }

        vector<uint64>::iterator lower = lower_bound(cellIDs_uqe.begin(), cellIDs_uqe.end(), cellID);
        uint64 gridID = lower - cellIDs_uqe.begin();
        gridDataIDs[gridID].push_back(i);
    }

    // construct index
    index.resize(gridSize);
    ordered2DataID.resize(dataSize);
    data2OrderedID.resize(dataSize);
    ordered2GridID.resize(dataSize);
    uint orderedID = 0;
    for (uint i = 0; i < gridSize; i++) {
        index[i].orderedID_min = orderedID;
        for (uint j = 0; j < gridDataIDs[i].size(); j++) {
            ordered2DataID[orderedID] = gridDataIDs[i][j];
            data2OrderedID[gridDataIDs[i][j]] = orderedID;
            ordered2GridID[orderedID] = i;
            orderedID++;
        }
        index[i].orderedID_max = orderedID - 1;
    }
}

__device__ int findGridByCellID(const uint cellID, const uint gridSize, const uint *d_grid2CellID) {
    int l = 0, r = gridSize - 1;
    int mid = (l + r) / 2;
    while (l <= r) {
        if (d_grid2CellID[mid] < cellID) {
            l = mid + 1;
        } else if (d_grid2CellID[mid] > cellID) {
            r = mid - 1;
        } else if (d_grid2CellID[mid] == cellID) {
            return mid;
        }
        mid = (l + r) / 2;
    }
    return gridSize + 1;
}

__device__ bool gpuInEpsilon(
    const uint orderedID0,
    const uint orderedID1,
    const float epsilonPow,
    const uint dataSize,
    const float *d_dataPoints,
    const uint *d_ordered2DataID) {
    uint id0 = d_ordered2DataID[orderedID0];
    uint id1 = d_ordered2DataID[orderedID1];
    return (pow(d_dataPoints[id0] - d_dataPoints[id1], 2) + pow(d_dataPoints[dataSize + id0] - d_dataPoints[dataSize + id1], 2)) <= epsilonPow;
}

__global__ void gpuCalcGlobal(
    const uint dataSize,
    const uint nCells1,
    const uint gridSize,
    const float epsilonPow,
    const float *d_dataPoints,
    const uint *d_ordered2DataID,
    const uint *d_ordered2GridID,
    const uint *d_grid2CellID,
    const Grid *d_index,
    uint *d_cnt,
    uint *d_orderedIDKey,
    uint *d_orderedIDValue) {
    uint orderedID = blockIdx.x * blockDim.x + threadIdx.x;
    if (orderedID >= dataSize) return;
    // uint dataID = d_ordered2DataID[orderedID];

    uint gridID = d_ordered2GridID[orderedID];
    uint cellID = d_grid2CellID[gridID];
    uint newCellID = 0;

    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) {
                gridID = d_ordered2GridID[orderedID];
            } else {
                newCellID = cellID + (i * nCells1 + j);
                // or a cell2GridID map
                gridID = findGridByCellID(newCellID, gridSize, d_grid2CellID);
            }

            if (gridID < gridSize) {
                for (uint k = d_index[gridID].orderedID_min; k <= d_index[gridID].orderedID_max; k++) {
                    if (orderedID == k) {
                        continue;
                    }
                    if (gpuInEpsilon(orderedID, k, epsilonPow, dataSize, d_dataPoints, d_ordered2DataID)) {
                        uint idx = atomicAdd(d_cnt, int(1));
                        d_orderedIDKey[idx] = orderedID;
                        d_orderedIDValue[idx] = k;
                    }
                }
            }
        }
    }
}

float HybridDBSCAN::constructGPUResultSet() {
    float gpu_elapsed_time_ms;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    float *d_dataPoints;
    hipMalloc((void **)&d_dataPoints, sizeof(float) * 2 * dataSize);
    hipMemcpy(d_dataPoints, dataPoints[0].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_dataPoints + dataSize, dataPoints[1].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);

    uint *d_ordered2DataID, *d_ordered2GridID, *d_grid2CellID;
    hipMalloc((void **)&d_ordered2DataID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_ordered2GridID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_grid2CellID, sizeof(uint) * dataSize);
    hipMemcpy(d_ordered2DataID, ordered2DataID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_ordered2GridID, ordered2GridID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_grid2CellID, grid2CellID.data(), sizeof(uint) * gridSize, hipMemcpyHostToDevice);

    Grid *d_index;
    hipMalloc((void **)&d_index, sizeof(Grid) * dataSize);
    hipMemcpy(d_index, index.data(), sizeof(Grid) * dataSize, hipMemcpyHostToDevice);

    uint *d_orderedIDKey;    //key
    uint *d_orderedIDValue;  //value
    hipMalloc((void **)&d_orderedIDKey, sizeof(uint) * GPU_BUFFER_SIZE);
    hipMalloc((void **)&d_orderedIDValue, sizeof(uint) * GPU_BUFFER_SIZE);
    hipHostMalloc((void **)&orderedIDKey, sizeof(uint) * GPU_BUFFER_SIZE);
    hipHostMalloc((void **)&orderedIDValue, sizeof(uint) * GPU_BUFFER_SIZE);

    uint *d_cnt;
    hipMalloc((void **)&d_cnt, sizeof(uint));

    dim3 dimGrid(ceil((double)dataSize / (double)blockSize));
    dim3 dimBlock(blockSize);

    gpuCalcGlobal<<<dimGrid, dimBlock>>>(dataSize, nCells[1], gridSize, epsilonPow, d_dataPoints, d_ordered2DataID, d_ordered2GridID, d_grid2CellID, d_index, d_cnt, d_orderedIDKey, d_orderedIDValue);
    hipDeviceSynchronize();

    neighborsCnt = 0;
    hipMemcpy(&neighborsCnt, d_cnt, sizeof(uint), hipMemcpyDeviceToHost);

    // sort gpuResultSet
    device_ptr<uint> d_keyPtr(d_orderedIDKey);
    device_ptr<uint> d_valuePtr(d_orderedIDValue);

    try {
        sort_by_key(d_keyPtr, d_keyPtr + neighborsCnt, d_valuePtr);
    } catch (std::bad_alloc &e) {
        fprintf(stderr, "Error: Ran out of memory while sorting.\n");
        exit(-1);
    }

    hipMemcpy(raw_pointer_cast(orderedIDKey), raw_pointer_cast(d_keyPtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);
    hipMemcpy(raw_pointer_cast(orderedIDValue), raw_pointer_cast(d_valuePtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);

    hipFree(d_dataPoints);
    hipFree(d_ordered2DataID);
    hipFree(d_ordered2GridID);
    hipFree(d_grid2CellID);
    hipFree(d_index);
    hipFree(d_cnt);
    hipFree(d_orderedIDKey);
    hipFree(d_orderedIDValue);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, end);

    return gpu_elapsed_time_ms;
}

void HybridDBSCAN::constructNeighborTable() {
    auto keyData_uqe = vector<KeyData>();
    keyData_uqe.push_back(KeyData(orderedIDKey[0], 0));
    for (uint i = 1; i < neighborsCnt; i++) {
        if (orderedIDKey[i] != orderedIDKey[i - 1]) {
            keyData_uqe.push_back(KeyData(orderedIDKey[i], i));
        }
    }

    neighborTables.resize(dataSize);
    uint key = 0;
    for (uint i = 0; i < keyData_uqe.size(); i++) {
        key = keyData_uqe[i].key;
        neighborTables[key].valueIdx_min = keyData_uqe[i].pos;
        if (i == keyData_uqe.size() - 1) {
            neighborTables[key].valueIdx_max = neighborsCnt - 1;
        } else {
            neighborTables[key].valueIdx_max = keyData_uqe[i + 1].pos - 1;
        }
    }
}

void HybridDBSCAN::DBSCANwithNeighborTable() {
    auto neighbors = vector<uint>();
    int clusterID = 0, orderedID = -1;

    for (uint i = 0; i < dataSize; i++) {
        orderedID = data2OrderedID[i];

        if (clusterIDs[i] != HybridDBSCAN::UNVISITED) {
            continue;
        }
        neighbors.clear();

        uint size = neighborTables[orderedID].valueIdx_max - neighborTables[orderedID].valueIdx_min + 1;

        if ((size + 1) < minpts) {
            // if (size < minpts) {
            clusterIDs[i] = HybridDBSCAN::NOISE;
        } else {
            clusterIDs[i] = ++clusterID;
            neighbors.resize(size);
            for (uint j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
                neighbors[j - neighborTables[orderedID].valueIdx_min] = orderedIDValue[j];
            }

            while (neighbors.size() != 0) {
                uint pOrderedID = neighbors.back();
                uint p = ordered2DataID[pOrderedID];

                if (clusterIDs[p] == HybridDBSCAN::UNVISITED) {
                    uint newSize = neighborTables[pOrderedID].valueIdx_max - neighborTables[pOrderedID].valueIdx_min + 1;
                    if ((newSize + 1) >= minpts) {
                        // if (newSize >= minpts) {
                        neighbors.resize(size + newSize);
                        for (uint j = neighborTables[pOrderedID].valueIdx_min; j <= neighborTables[pOrderedID].valueIdx_max; j++) {
                            neighbors[size + j - neighborTables[pOrderedID].valueIdx_min] = orderedIDValue[j];
                        }
                    }
                }

                if (clusterIDs[p] == HybridDBSCAN::UNVISITED || clusterIDs[p] == HybridDBSCAN::NOISE) {
                    clusterIDs[p] = clusterID;
                }

                neighbors.pop_back();
            }
        }
    }
}

void HybridDBSCAN::run() {
    float start = omp_get_wtime();
    constructIndex();
    float gpu_elapsed_time_ms = constructGPUResultSet();
    constructNeighborTable();
    // debug_printNeighborTable();
    DBSCANwithNeighborTable();
    float end = omp_get_wtime();
    float elapsed_time_ms = (end - start) * 1000;
    printf("Time elapsed on Hybrid-DBSCAN: %f ms; gpu_elapsed_time: %f ms\n", elapsed_time_ms, gpu_elapsed_time_ms);
}

void HybridDBSCAN::print(const string &outFile) {
    ofstream out(outFile);
    if (out.is_open()) {
        out << "x,y,clusterID\n";
        char buffer[200];
        for (uint i = 0; i < dataSize; i++) {
            sprintf(buffer, "%5.15lf,%5.15lf,%d\n", dataPoints[0][i], dataPoints[1][i], clusterIDs[i]);
            out << buffer;
        }
        out.close();
    } else {
        cout << "Unable to open file " << outFile << endl;
    }
}

vector<uint> HybridDBSCAN::debug_getNeighbors(const uint &id) {
    uint orderedID = data2OrderedID[id];
    auto neighbors = vector<uint>();
    for (int i = neighborTables[orderedID].valueIdx_min; i <= neighborTables[orderedID].valueIdx_max; i++) {
        neighbors.push_back(ordered2DataID[orderedIDValue[i]]);
    }
    sort(neighbors.begin(), neighbors.end());
    return neighbors;
}

void HybridDBSCAN::debug_printNeighborTable() {
    for (int i = 0; i < dataSize; i++) {
        uint orderedID = data2OrderedID[i];
        printf("\ndataID = %d, neighbors's dataID: ", i);
        for (int j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
            printf("%d, ", ordered2DataID[orderedIDValue[j]]);
        }
    }
    printf("\n");
}