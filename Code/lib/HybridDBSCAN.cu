#include <algorithm>
#include <fstream>
#include <iostream>

#include "HybridDBSCAN.hpp"
#include "utils.hpp"

using std::cout;
using std::endl;
using std::ofstream;

HybridDBSCAN::HybridDBSCAN(
    const float epsilon,
    const uint minpts,
    DataPointsType dataPoints,
    uint dataSize,
    uint blockSize,
    uint NCHUNKS) : DBSCAN(epsilon, minpts, dataPoints, dataSize),
                    blockSize(blockSize),
                    NCHUNKS(NCHUNKS),
                    gridSize(0) {
    printf("start HybridDBSCAN::HybridDBSCAN\n");
    if ((dataSize % NCHUNKS) != 0) {
        fprintf(stderr, "Error: dataSize mod NCHUNKS != 0\n");
    }
    clusterIDs.resize(dataSize);
    for (int i = 0; i < dataSize; i++) {
        clusterIDs[i] = HybridDBSCAN::UNVISITED;
    }
    neighborTables.resize(dataSize);
    valuePtrs.resize(NCHUNKS);

#pragma omp parallel for
    for (int i = 0; i < GPU_STREAMS; i++) {
        CHECK(hipHostMalloc((void **)&orderedIDKeys[i], sizeof(uint) * GPU_BUFFER_SIZE));
        CHECK(hipHostMalloc((void **)&orderedIDValues[i], sizeof(uint) * GPU_BUFFER_SIZE));
    }

    for (int i = 0; i < GPU_STREAMS; i++) {
        neighborsCnts[i] = 0;
    }
    printf("finish HybridDBSCAN::HybridDBSCAN\n");
}

HybridDBSCAN::~HybridDBSCAN() {
#pragma omp parallel for
    for (int i = 0; i < GPU_STREAMS; i++) {
        hipFree(orderedIDKeys[i]);
        hipFree(orderedIDValues[i]);
    }
    for (int i = 0; i < NCHUNKS; i++) {
        free(valuePtrs[i]);
    }
}

void HybridDBSCAN::run() {
    printf("start HybridDBSCAN::run\n");

    auto start = system_clock::now();
    calcCells(minVals, maxVals, nCells, totalCells);
    constructIndex(
        minVals,
        maxVals,
        nCells,
        totalCells,
        gridSize,
        index,
        ordered2GridID,
        ordered2DataID,
        data2OrderedID,
        grid2CellID);

    constructResultSetAndNeighborTable(
        NCHUNKS,
        nCells,
        gridSize,
        ordered2GridID,
        ordered2DataID,
        grid2CellID,
        index,
        blockSize,
        neighborsCnts,
        orderedIDKeys,
        orderedIDValues,
        valuePtrs,
        neighborTables);

    /*
    float gpu_elapsed_time_ms[NCHUNKS];
#pragma omp parallel for schedule(static, 1) num_threads(GPU_STREAMS)
    for (uint i = 0; i < NCHUNKS; i++) {
        printf("in run: i = %d\n", i);
        uint streamID = i % GPU_STREAMS;
        neighborsCnts[streamID] = 0;
        gpu_elapsed_time_ms[i] = constructGPUResultSet(
            i,
            NCHUNKS,
            nCells,
            gridSize,
            ordered2GridID,
            ordered2DataID,
            grid2CellID,
            index,
            blockSize,
            neighborsCnts[streamID],
            orderedIDKeys[streamID],
            orderedIDValues[streamID]);

        valuePtrs[i] = (uint *)malloc(sizeof(uint) * neighborsCnts[streamID]);

        constructNeighborTable(
            orderedIDKeys[streamID],
            orderedIDValues[streamID],
            neighborsCnts[streamID],
            valuePtrs[i],
            neighborTables);
    }
    */

    DBSCANwithNeighborTable(
        data2OrderedID,
        ordered2DataID,
        neighborTables,
        clusterIDs);

    auto end = system_clock::now();
    auto duration = duration_cast<microseconds>(end - start);
    cout << "Time elapsed on Hybrid-DBSCAN: "
         << double(duration.count()) * microseconds::period::num / milliseconds::period::den
         << "ms" << endl;

    printf("finish HybridDBSCAN::run\n");
}

void HybridDBSCAN::print(const string &outFile) {
    ofstream out(outFile);
    if (out.is_open()) {
        out << "x,y,clusterID\n";
        char buffer[200];
        for (uint i = 0; i < dataSize; i++) {
            sprintf(buffer, "%5.15lf,%5.15lf,%d\n", dataPoints[0][i], dataPoints[1][i], clusterIDs[i]);
            out << buffer;
        }
        out.close();
    } else {
        cout << "Unable to open file " << outFile << endl;
    }
}

vector<uint> HybridDBSCAN::debug_getNeighbors(const uint &id) {
    uint orderedID = data2OrderedID[id];
    auto neighbors = vector<uint>();
    for (int i = neighborTables[orderedID].valueIdx_min; i <= neighborTables[orderedID].valueIdx_max; i++) {
        neighbors.push_back(ordered2DataID[neighborTables[orderedID].values[i]]);
    }
    sort(neighbors.begin(), neighbors.end());
    return neighbors;
}

void HybridDBSCAN::debug_printNeighborTable() {
    printf("in HybridDBSCAN::debug_printNeighborTable\n");

    for (uint i = 0; i < dataSize; i++) {
        uint orderedID = data2OrderedID[i];
        printf("\ndataID = %u, orderedID = %u, valueIdx_min = %d, valueIdx_max = %d\n", i, orderedID, neighborTables[orderedID].valueIdx_min, neighborTables[orderedID].valueIdx_max);
        for (int j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
            printf("%d, ", ordered2DataID[neighborTables[orderedID].values[j]]);
        }
        printf("\n");
    }
    printf("\n");
    printf("finish HybridDBSCAN::debug_printNeighborTable\n");
}