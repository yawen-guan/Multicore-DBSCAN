#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <fstream>
#include <iostream>
#include <parallel/algorithm>

#include "HybridDBSCAN.cuh"

using std::ceil;
using std::cout;
using std::endl;
using std::iostream;
using std::lower_bound;
using std::max;
using std::min;
using std::ofstream;
using thrust::device_vector;

HybridDBSCAN::HybridDBSCAN(
    const float epsilon,
    const uint minpts,
    DataPointsType dataPoints,
    uint dataSize,
    uint blockSize) : DBSCAN(epsilon, minpts, dataPoints, dataSize),
                      blockSize(blockSize),
                      gridSize(0) {
    clusterIDs.resize(dataSize);
    for (int i = 0; i < dataSize; i++) {
        clusterIDs[i] = HybridDBSCAN::UNVISITED;
    }
}

void HybridDBSCAN::run() {
    float start = omp_get_wtime();
    constructIndex(
        minVals,
        maxVals,
        nCells,
        totalCells,
        gridSize,
        index,
        ordered2GridID,
        ordered2DataID,
        data2OrderedID,
        grid2CellID);
    float gpu_elapsed_time_ms = constructGPUResultSet(
        nCells,
        gridSize,
        ordered2GridID,
        ordered2DataID,
        grid2CellID,
        index,
        blockSize,
        neighborsCnt,
        orderedIDKey,
        orderedIDValue);
    constructNeighborTable(
        orderedIDKey,
        orderedIDValue,
        neighborsCnt,
        neighborTables);
    DBSCANwithNeighborTable(
        data2OrderedID,
        ordered2DataID,
        orderedIDValue,
        neighborTables,
        clusterIDs);
    float end = omp_get_wtime();
    float elapsed_time_ms = (end - start) * 1000;
    printf("Time elapsed on Hybrid-DBSCAN: %f ms; gpu_elapsed_time: %f ms\n", elapsed_time_ms, gpu_elapsed_time_ms);
}

void HybridDBSCAN::print(const string &outFile) {
    ofstream out(outFile);
    if (out.is_open()) {
        out << "x,y,clusterID\n";
        char buffer[200];
        for (uint i = 0; i < dataSize; i++) {
            sprintf(buffer, "%5.15lf,%5.15lf,%d\n", dataPoints[0][i], dataPoints[1][i], clusterIDs[i]);
            out << buffer;
        }
        out.close();
    } else {
        cout << "Unable to open file " << outFile << endl;
    }
}

vector<uint> HybridDBSCAN::debug_getNeighbors(const uint &id) {
    uint orderedID = data2OrderedID[id];
    auto neighbors = vector<uint>();
    for (int i = neighborTables[orderedID].valueIdx_min; i <= neighborTables[orderedID].valueIdx_max; i++) {
        neighbors.push_back(ordered2DataID[orderedIDValue[i]]);
    }
    sort(neighbors.begin(), neighbors.end());
    return neighbors;
}

void HybridDBSCAN::debug_printNeighborTable() {
    for (int i = 0; i < dataSize; i++) {
        uint orderedID = data2OrderedID[i];
        printf("\ndataID = %d, neighbors's dataID: ", i);
        for (int j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
            printf("%d, ", ordered2DataID[orderedIDValue[j]]);
        }
    }
    printf("\n");
}