#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>
#include <parallel/algorithm>

#include "DBSCAN.hpp"

using std::ceil;
using std::copy;
using std::cout;
using std::endl;
using std::iostream;
using std::lower_bound;
using std::max;
using std::min;
using std::ofstream;
using thrust::device_ptr;
using thrust::device_vector;
using thrust::raw_pointer_cast;
using thrust::sort_by_key;

DBSCAN::DBSCAN(
    const float epsilon,
    const uint minpts,
    DataPointsType dataPoints,
    uint dataSize) : epsilon(epsilon),
                     epsilonPow(pow(epsilon, 2)),
                     minpts(minpts),
                     dataPoints(dataPoints),
                     dataSize(dataSize) {
}

DBSCAN::~DBSCAN() {
}

void DBSCAN::calcCells(
    array<float, 2> &minVals,
    array<float, 2> &maxVals,
    array<uint, 2> &nCells,
    uint64 &totalCells) {
    // minVals, maxVals
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] = dataPoints[dim][0];
        maxVals[dim] = dataPoints[dim][0];
    }
    for (uint dim = 0; dim < 2; dim++) {
        for (uint i = 0; i < dataSize; i++) {
            minVals[dim] = min(minVals[dim], dataPoints[dim][i]);
            maxVals[dim] = max(maxVals[dim], dataPoints[dim][i]);
        }
    }
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] -= epsilon;
        maxVals[dim] += epsilon;
    }
    // nCells
    totalCells = 1;
    for (uint dim = 0; dim < 2; dim++) {
        nCells[dim] = ceil((maxVals[dim] - minVals[dim]) / epsilon);
        totalCells *= nCells[dim];
    }
}

void DBSCAN::constructIndex(
    const array<float, 2> &minVals,
    const array<float, 2> &maxVals,
    const array<uint, 2> &nCells,
    const uint64 &totalCells,
    uint &gridSize,
    vector<Grid> &index,
    vector<uint> &ordered2GridID,
    vector<uint> &ordered2DataID,
    vector<uint> &data2OrderedID,
    vector<uint> &grid2CellID) {
    printf("start DBSCAN::constructIndex\n");

    // calculate cellIDs of grid cells
    auto cellIDs_dup = vector<uint64>();
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        cellIDs_dup.push_back(cellID);
    }
    omp_set_num_threads(NUM_THREADS);
    __gnu_parallel::sort(cellIDs_dup.begin(), cellIDs_dup.end());
    // std::sort(cellIDs_dup.begin(), cellIDs_dup.end());
    auto cellIDs_uqe = vector<uint64>();
    cellIDs_uqe.push_back(cellIDs_dup[0]);
    for (uint i = 1; i < cellIDs_dup.size(); i++) {
        if (cellIDs_dup[i] != cellIDs_dup[i - 1]) {
            cellIDs_uqe.push_back(cellIDs_dup[i]);
        }
    }

    grid2CellID.resize(cellIDs_uqe.size());
    for (uint i = 0; i < cellIDs_uqe.size(); i++) {
        grid2CellID[i] = cellIDs_uqe[i];
    }

    // get dataIDs per non empty cell(aka. grid)
    gridSize = cellIDs_uqe.size();
    auto gridDataIDs = vector<vector<uint64>>(gridSize);  // dataID per cell
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        if (cellID > totalCells) {
            fprintf(stderr, "ERROR: LinearID = %lu > totalCells = %lu\n", cellID, totalCells);
        }

        vector<uint64>::iterator lower = lower_bound(cellIDs_uqe.begin(), cellIDs_uqe.end(), cellID);
        uint64 gridID = lower - cellIDs_uqe.begin();
        gridDataIDs[gridID].push_back(i);
    }

    // construct index
    index.resize(gridSize);
    ordered2DataID.resize(dataSize);
    data2OrderedID.resize(dataSize);
    ordered2GridID.resize(dataSize);
    uint orderedID = 0;
    for (uint i = 0; i < gridSize; i++) {
        index[i].orderedID_min = orderedID;
        for (uint j = 0; j < gridDataIDs[i].size(); j++) {
            ordered2DataID[orderedID] = gridDataIDs[i][j];
            data2OrderedID[gridDataIDs[i][j]] = orderedID;
            ordered2GridID[orderedID] = i;
            orderedID++;
        }
        index[i].orderedID_max = orderedID - 1;
    }

    printf("finish DBSCAN::constructIndex\n");
}

__device__ int findGridByCellID(const uint cellID, const uint gridSize, const uint *d_grid2CellID) {
    int l = 0, r = gridSize - 1;
    int mid = (l + r) / 2;
    while (l <= r) {
        if (d_grid2CellID[mid] < cellID) {
            l = mid + 1;
        } else if (d_grid2CellID[mid] > cellID) {
            r = mid - 1;
        } else if (d_grid2CellID[mid] == cellID) {
            return mid;
        }
        mid = (l + r) / 2;
    }
    return gridSize + 1;
}

__device__ bool gpuInEpsilon(
    const uint orderedID0,
    const uint orderedID1,
    const float epsilonPow,
    const uint dataSize,
    const float *d_dataPoints,
    const uint *d_ordered2DataID) {
    uint id0 = d_ordered2DataID[orderedID0];
    uint id1 = d_ordered2DataID[orderedID1];
    return (pow(d_dataPoints[id0] - d_dataPoints[id1], 2) + pow(d_dataPoints[dataSize + id0] - d_dataPoints[dataSize + id1], 2)) <= epsilonPow;
}

__global__ void gpuCalcGlobal(
    const uint chunkID,
    const uint NCHUNKS,
    const uint chunkSize,
    const uint dataSize,
    const uint nCells1,
    const uint gridSize,
    const float epsilonPow,
    const float *d_dataPoints,
    const uint *d_ordered2DataID,
    const uint *d_ordered2GridID,
    const uint *d_grid2CellID,
    const Grid *d_index,
    uint *d_cnt,
    uint *d_orderedIDKey,
    uint *d_orderedIDValue) {
    uint globalID = blockIdx.x * blockDim.x + threadIdx.x;
    // if (globalID >= (dataSize / NCHUNKS)) return;
    if (globalID >= chunkSize) return;

    uint orderedID = globalID * NCHUNKS + chunkID;
    // uint dataID = d_ordered2DataID[orderedID];
    // printf("dataID = %u, orderedID = %u, globalID = %u, chunkID = %u\n", dataID, orderedID, globalID, chunkID);

    uint gridID = d_ordered2GridID[orderedID];
    uint cellID = d_grid2CellID[gridID];
    uint newCellID = 0;

    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) {
                gridID = d_ordered2GridID[orderedID];
            } else {
                newCellID = cellID + (i * nCells1 + j);
                // or a cell2GridID map
                gridID = findGridByCellID(newCellID, gridSize, d_grid2CellID);
            }

            if (gridID < gridSize) {
                for (uint k = d_index[gridID].orderedID_min; k <= d_index[gridID].orderedID_max; k++) {
                    if (orderedID == k) {
                        continue;
                    }
                    if (gpuInEpsilon(orderedID, k, epsilonPow, dataSize, d_dataPoints, d_ordered2DataID)) {
                        uint idx = atomicAdd(d_cnt, int(1));
                        d_orderedIDKey[idx] = orderedID;
                        d_orderedIDValue[idx] = k;
                    }
                }
            }
        }
    }
}

float DBSCAN::constructGPUResultSet(
    const uint &chunkID,
    const uint &NCHUNKS,
    const array<uint, 2> &nCells,
    const uint &gridSize,
    const vector<uint> &ordered2GridID,
    const vector<uint> &ordered2DataID,
    const vector<uint> &grid2CellID,
    const vector<Grid> &index,
    const uint &blockSize,
    uint &neighborsCnt,
    uint *&orderedIDKey,
    uint *&orderedIDValue) {
    float gpu_elapsed_time_ms;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    float *d_dataPoints;
    CHECK(hipMalloc((void **)&d_dataPoints, sizeof(float) * 2 * dataSize));
    CHECK(hipMemcpy(d_dataPoints, dataPoints[0].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_dataPoints + dataSize, dataPoints[1].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice));

    uint *d_ordered2DataID, *d_ordered2GridID, *d_grid2CellID;
    hipMalloc((void **)&d_ordered2DataID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_ordered2GridID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_grid2CellID, sizeof(uint) * dataSize);
    hipMemcpy(d_ordered2DataID, ordered2DataID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_ordered2GridID, ordered2GridID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_grid2CellID, grid2CellID.data(), sizeof(uint) * gridSize, hipMemcpyHostToDevice);

    Grid *d_index;
    hipMalloc((void **)&d_index, sizeof(Grid) * dataSize);
    hipMemcpy(d_index, index.data(), sizeof(Grid) * dataSize, hipMemcpyHostToDevice);

    uint *d_orderedIDKey;
    uint *d_orderedIDValue;
    CHECK(hipMalloc((void **)&d_orderedIDKey, sizeof(uint) * GPU_BUFFER_SIZE));
    CHECK(hipMalloc((void **)&d_orderedIDValue, sizeof(uint) * GPU_BUFFER_SIZE));

    uint *d_cnt;
    CHECK(hipMalloc((void **)&d_cnt, sizeof(uint)));
    uint chunkSize = dataSize / NCHUNKS;

    dim3 dimGrid(ceil((double)dataSize / (double)NCHUNKS / (double)blockSize));
    dim3 dimBlock(blockSize);

    gpuCalcGlobal<<<dimGrid, dimBlock>>>(chunkID, NCHUNKS, chunkSize, dataSize, nCells[1], gridSize, epsilonPow, d_dataPoints, d_ordered2DataID, d_ordered2GridID, d_grid2CellID, d_index, d_cnt, d_orderedIDKey, d_orderedIDValue);
    hipDeviceSynchronize();

    neighborsCnt = 0;
    hipMemcpy(&neighborsCnt, d_cnt, sizeof(uint), hipMemcpyDeviceToHost);

    // sort gpuResultSet
    device_ptr<uint> d_keyPtr(d_orderedIDKey);
    device_ptr<uint> d_valuePtr(d_orderedIDValue);

    try {
        sort_by_key(d_keyPtr, d_keyPtr + neighborsCnt, d_valuePtr);
    } catch (std::bad_alloc &e) {
        fprintf(stderr, "Error: Ran out of memory while sorting.\n");
        exit(-1);
    }

    hipMemcpy(raw_pointer_cast(orderedIDKey), raw_pointer_cast(d_keyPtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);
    hipMemcpy(raw_pointer_cast(orderedIDValue), raw_pointer_cast(d_valuePtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);

    hipFree(d_dataPoints);
    hipFree(d_ordered2DataID);
    hipFree(d_ordered2GridID);
    hipFree(d_grid2CellID);
    hipFree(d_index);
    hipFree(d_cnt);
    hipFree(d_orderedIDKey);
    hipFree(d_orderedIDValue);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, end);

    return gpu_elapsed_time_ms;
}

void DBSCAN::constructNeighborTable(
    const uint *orderedIDKey,
    const uint *orderedIDValue,
    const uint &neighborsCnt,
    uint *&valuePtr,
    vector<NeighborTable> &neighborTables) {
    copy(orderedIDValue, orderedIDValue + neighborsCnt, valuePtr);

    auto keyData_uqe = vector<KeyData>();
    keyData_uqe.push_back(KeyData(orderedIDKey[0], 0));
    for (uint i = 1; i < neighborsCnt; i++) {
        if (orderedIDKey[i] != orderedIDKey[i - 1]) {
            keyData_uqe.push_back(KeyData(orderedIDKey[i], i));
        }
    }

    // neighborTables.resize(dataSize);
    uint key = 0;
    for (uint i = 0; i < keyData_uqe.size(); i++) {
        key = keyData_uqe[i].key;
        neighborTables[key].values = valuePtr;
        neighborTables[key].valueIdx_min = keyData_uqe[i].pos;
        if (i == keyData_uqe.size() - 1) {
            neighborTables[key].valueIdx_max = neighborsCnt - 1;
        } else {
            neighborTables[key].valueIdx_max = keyData_uqe[i + 1].pos - 1;
        }
    }
}

void DBSCAN::constructResultSetAndNeighborTable(
    const uint &NCHUNKS,
    const array<uint, 2> &nCells,
    const uint &gridSize,
    const vector<uint> &ordered2GridID,
    const vector<uint> &ordered2DataID,
    const vector<uint> &grid2CellID,
    const vector<Grid> &index,
    const uint &blockSize,
    uint neighborsCnts[GPU_STREAMS],
    uint *orderedIDKeys[GPU_STREAMS],
    uint *orderedIDValues[GPU_STREAMS],
    vector<uint *> &valuePtrs,
    vector<NeighborTable> &neighborTables) {
    printf("start DBSCAN::constructResultSetAndNeighborTable\n");

    float gpu_elapsed_time_ms;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    float *d_dataPoints;
    hipMalloc((void **)&d_dataPoints, sizeof(float) * 2 * dataSize);
    hipMemcpy(d_dataPoints, dataPoints[0].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_dataPoints + dataSize, dataPoints[1].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);

    uint *d_ordered2DataID, *d_ordered2GridID, *d_grid2CellID;
    hipMalloc((void **)&d_ordered2DataID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_ordered2GridID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_grid2CellID, sizeof(uint) * dataSize);
    hipMemcpy(d_ordered2DataID, ordered2DataID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_ordered2GridID, ordered2GridID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_grid2CellID, grid2CellID.data(), sizeof(uint) * gridSize, hipMemcpyHostToDevice);

    Grid *d_index;
    hipMalloc((void **)&d_index, sizeof(Grid) * dataSize);
    hipMemcpy(d_index, index.data(), sizeof(Grid) * dataSize, hipMemcpyHostToDevice);

    uint *d_orderedIDKeys[GPU_STREAMS];
    uint *d_orderedIDValues[GPU_STREAMS];
    for (uint i = 0; i < GPU_STREAMS; i++) {
        CHECK(hipMalloc((void **)&d_orderedIDKeys[i], sizeof(uint) * GPU_BUFFER_SIZE));
        CHECK(hipMalloc((void **)&d_orderedIDValues[i], sizeof(uint) * GPU_BUFFER_SIZE));
    }

    uint *d_cnt;
    CHECK(hipMalloc((void **)&d_cnt, sizeof(uint) * GPU_STREAMS));

    uint chunkSize = dataSize / NCHUNKS;

    hipStream_t stream[GPU_STREAMS];
    for (uint i = 0; i < GPU_STREAMS; i++) {
        hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
    }

// #pragma omp parallel for schedule(static, 1) num_threads(GPU_STREAMS)
#pragma omp parallel for schedule(static, 1) num_threads(GPU_STREAMS)
    for (uint i = 0; i < NCHUNKS; i++) {
        uint chunkID = i;
        int streamID = omp_get_thread_num();
        printf("chunkID = %u, streamID = %d\n", chunkID, streamID);

        neighborsCnts[streamID] = 0;
        CHECK(hipMemcpyAsync(&d_cnt[streamID], &neighborsCnts[streamID], sizeof(uint), hipMemcpyHostToDevice, stream[streamID]));

        dim3 dimGrid(ceil((double)chunkSize / (double)blockSize));
        dim3 dimBlock(blockSize);
        gpuCalcGlobal<<<dimGrid, dimBlock, 0, stream[streamID]>>>(chunkID, NCHUNKS, chunkSize, dataSize, nCells[1], gridSize, epsilonPow, d_dataPoints, d_ordered2DataID, d_ordered2GridID, d_grid2CellID, d_index, &d_cnt[chunkID], d_orderedIDKeys[streamID], d_orderedIDValues[streamID]);

        CHECK(hipMemcpyAsync(&neighborsCnts[streamID], &d_cnt[streamID], sizeof(uint), hipMemcpyDeviceToHost, stream[streamID]));

        // sort gpuResultSet
        device_ptr<uint> d_keyPtr(d_orderedIDKeys[streamID]);
        device_ptr<uint> d_valuePtr(d_orderedIDValues[streamID]);

        try {
            sort_by_key(thrust::cuda::par.on(stream[streamID]), d_keyPtr, d_keyPtr + neighborsCnts[streamID], d_valuePtr);
        } catch (std::bad_alloc &e) {
            fprintf(stderr, "Error: Ran out of memory while sorting.\n");
            exit(-1);
        }

        hipMemcpyAsync(raw_pointer_cast(orderedIDKeys[streamID]), raw_pointer_cast(d_keyPtr), sizeof(uint) * neighborsCnts[streamID], hipMemcpyDeviceToHost, stream[streamID]);
        hipMemcpyAsync(raw_pointer_cast(orderedIDValues[streamID]), raw_pointer_cast(d_valuePtr), sizeof(uint) * neighborsCnts[streamID], hipMemcpyDeviceToHost, stream[streamID]);

        hipStreamSynchronize(stream[streamID]);

        valuePtrs[chunkID] = new uint[neighborsCnts[streamID]];
        constructNeighborTable(orderedIDKeys[streamID], orderedIDValues[streamID], neighborsCnts[streamID], valuePtrs[chunkID], neighborTables);
    }

    hipFree(d_dataPoints);
    hipFree(d_ordered2DataID);
    hipFree(d_ordered2GridID);
    hipFree(d_grid2CellID);
    hipFree(d_index);
    hipFree(d_cnt);
    for (uint i = 0; i < GPU_STREAMS; i++) {
        hipFree(d_orderedIDKeys[i]);
        hipFree(d_orderedIDValues[i]);
    }

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, end);

    for (int i = 0; i < GPU_STREAMS; i++) {
        CHECK(hipStreamDestroy(stream[i]));
    }

    printf("finish DBSCAN::constructResultSetAndNeighborTable\n");
    // return gpu_elapsed_time_ms;
}

void DBSCAN::DBSCANwithNeighborTable(
    const vector<uint> &data2OrderedID,
    const vector<uint> &ordered2DataID,
    const vector<NeighborTable> &neighborTables,
    vector<int> &clusterIDs) {
    auto neighbors = vector<uint>();
    int clusterID = 0, orderedID = -1, size = 0, newSize = 0;

    for (uint i = 0; i < dataSize; i++) {
        orderedID = data2OrderedID[i];

        if (clusterIDs[i] != DBSCAN::UNVISITED) {
            continue;
        }
        neighbors.clear();

        if (neighborTables[orderedID].valueIdx_min == -1 || neighborTables[orderedID].valueIdx_max == -1) {
            size = 0;
        } else {
            size = neighborTables[orderedID].valueIdx_max - neighborTables[orderedID].valueIdx_min + 1;
        }

        if ((size + 1) < minpts) {
            clusterIDs[i] = DBSCAN::NOISE;
        } else {
            clusterIDs[i] = ++clusterID;
            neighbors.resize(size);
            for (uint j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
                neighbors[j - neighborTables[orderedID].valueIdx_min] = neighborTables[orderedID].values[j];  // orderedIDValue[j];
            }

            while (neighbors.size() != 0) {
                uint pOrderedID = neighbors.back();
                uint p = ordered2DataID[pOrderedID];

                if (clusterIDs[p] == DBSCAN::UNVISITED) {
                    if (neighborTables[pOrderedID].valueIdx_min == -1 || neighborTables[pOrderedID].valueIdx_max == -1) {
                        newSize = 0;
                    } else {
                        newSize = neighborTables[pOrderedID].valueIdx_max - neighborTables[pOrderedID].valueIdx_min + 1;
                    }

                    if ((newSize + 1) >= minpts) {
                        neighbors.resize(size + newSize);
                        for (uint j = neighborTables[pOrderedID].valueIdx_min; j <= neighborTables[pOrderedID].valueIdx_max; j++) {
                            neighbors[size + j - neighborTables[pOrderedID].valueIdx_min] = neighborTables[pOrderedID].values[j];  // orderedIDValue[j];
                        }
                    }
                }

                if (clusterIDs[p] == DBSCAN::UNVISITED || clusterIDs[p] == DBSCAN::NOISE) {
                    clusterIDs[p] = clusterID;
                }

                neighbors.pop_back();
            }
        }
    }
}