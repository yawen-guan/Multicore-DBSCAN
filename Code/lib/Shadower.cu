#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>
#include <parallel/algorithm>

#include "Shadower.cuh"

using std::ceil;
using std::copy;
using std::cout;
using std::endl;
using std::iostream;
using std::lower_bound;
using std::max;
using std::min;
using std::ofstream;
using thrust::device_ptr;
using thrust::device_vector;
using thrust::raw_pointer_cast;
using thrust::sort_by_key;

Shadower::Shadower(
    const float epsilon,
    const uint minpts,
    DataPointsType dataPoints,
    uint dataSize,
    uint blockSize, 
    vector<vector<int>> clusterIDsArray, 
    vector<uint> pointIDs_shadow, 
    vector<PointChunkLookup> pointChunkMapping) : epsilon(epsilon),
                      epsilonPow(pow(epsilon, 2)),
                      minpts(minpts),
                      dataPoints(dataPoints),
                      dataSize(dataSize),
                      blockSize(blockSize),
                      gridSize(0),
                      orderedIDKey(nullptr),
                      orderedIDValue(nullptr),
                      neighborsCnt(0),
                      clusterIDsArray(clusterIDsArray), 
                      pointIDs_shadow(pointIDs_shadow), 
                      pointChunkMapping(pointChunkMapping) {
    clusterIDs.resize(dataSize);
    for (int i = 0; i < dataSize; i++) {
        clusterIDs[i] = Shadower::UNVISITED;
    }
}

void Shadower::calcCells() {
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] = dataPoints[dim][0];
        maxVals[dim] = dataPoints[dim][0];
    }
    for (uint dim = 0; dim < 2; dim++) {
        for (uint i = 0; i < dataSize; i++) {
            minVals[dim] = min(minVals[dim], dataPoints[dim][i]);
            maxVals[dim] = max(maxVals[dim], dataPoints[dim][i]);
        }
    }
    for (uint dim = 0; dim < 2; dim++) {
        minVals[dim] -= epsilon;
        maxVals[dim] += epsilon;
    }
    totalCells = 1;
    for (uint dim = 0; dim < 2; dim++) {
        nCells[dim] = ceil((maxVals[dim] - minVals[dim]) / epsilon);
        totalCells *= nCells[dim];
    }
}

void Shadower::constructIndex() {
    calcCells();
    auto cellIDs_dup = vector<uint64>();
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        cellIDs_dup.push_back(cellID);
    }
    std::sort(cellIDs_dup.begin(), cellIDs_dup.end());
    auto cellIDs_uqe = vector<uint64>();
    cellIDs_uqe.push_back(cellIDs_dup[0]);
    for (uint i = 1; i < cellIDs_dup.size(); i++) {
        if (cellIDs_dup[i] != cellIDs_dup[i - 1]) {
            cellIDs_uqe.push_back(cellIDs_dup[i]);
        }
    }

    grid2CellID.resize(cellIDs_uqe.size());
    for (uint i = 0; i < cellIDs_uqe.size(); i++) {
        grid2CellID[i] = cellIDs_uqe[i];
    }
    gridSize = cellIDs_uqe.size();
    auto gridDataIDs = vector<vector<uint64>>(gridSize); 
    for (uint i = 0; i < dataSize; i++) {
        uint64 l0 = (dataPoints[0][i] - minVals[0]) / epsilon;
        uint64 l1 = (dataPoints[1][i] - minVals[1]) / epsilon;
        uint64 cellID = l0 * nCells[1] + l1;
        if (cellID > totalCells) {
            fprintf(stderr, "ERROR: LinearID = %lu > totalCells = %lu\n", cellID, totalCells);
        }

        vector<uint64>::iterator lower = lower_bound(cellIDs_uqe.begin(), cellIDs_uqe.end(), cellID);
        uint64 gridID = lower - cellIDs_uqe.begin();
        gridDataIDs[gridID].push_back(i);
    }

    index.resize(gridSize);
    ordered2DataID.resize(dataSize);
    data2OrderedID.resize(dataSize);
    ordered2GridID.resize(dataSize);
    uint orderedID = 0;
    for (uint i = 0; i < gridSize; i++) {
        index[i].orderedID_min = orderedID;
        for (uint j = 0; j < gridDataIDs[i].size(); j++) {
            ordered2DataID[orderedID] = gridDataIDs[i][j];
            data2OrderedID[gridDataIDs[i][j]] = orderedID;
            ordered2GridID[orderedID] = i;
            orderedID++;
        }
        index[i].orderedID_max = orderedID - 1;
    }
}

__device__ int findGridByCellID_s(const uint cellID, const uint gridSize, const uint *d_grid2CellID) {
    int l = 0, r = gridSize - 1;
    int mid = (l + r) / 2;
    while (l <= r) {
        if (d_grid2CellID[mid] < cellID) {
            l = mid + 1;
        } else if (d_grid2CellID[mid] > cellID) {
            r = mid - 1;
        } else if (d_grid2CellID[mid] == cellID) {
            return mid;
        }
        mid = (l + r) / 2;
    }
    return gridSize + 1;
}

__device__ bool gpuInEpsilon_s(
    const uint orderedID0,
    const uint orderedID1,
    const float epsilonPow,
    const uint dataSize,
    const float *d_dataPoints,
    const uint *d_ordered2DataID) {
    uint id0 = d_ordered2DataID[orderedID0];
    uint id1 = d_ordered2DataID[orderedID1];
    return (pow(d_dataPoints[id0] - d_dataPoints[id1], 2) + pow(d_dataPoints[dataSize + id0] - d_dataPoints[dataSize + id1], 2)) <= epsilonPow;
}

__global__ void gpuCalcGlobal_s(
    const uint dataSize,
    const uint nCells1,
    const uint gridSize,
    const float epsilonPow,
    const float *d_dataPoints,
    const uint *d_ordered2DataID,
    const uint *d_ordered2GridID,
    const uint *d_grid2CellID,
    const Grid *d_index,
    uint *d_cnt,
    uint *d_orderedIDKey,
    uint *d_orderedIDValue) {
    uint orderedID = blockIdx.x * blockDim.x + threadIdx.x;
    if (orderedID >= dataSize) return;

    uint gridID = d_ordered2GridID[orderedID];
    uint cellID = d_grid2CellID[gridID];
    uint newCellID = 0;

    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (i == 0 && j == 0) {
                gridID = d_ordered2GridID[orderedID];
            } else {
                newCellID = cellID + (i * nCells1 + j);
                gridID = findGridByCellID_s(newCellID, gridSize, d_grid2CellID);
            }

            if (gridID < gridSize) {
                for (uint k = d_index[gridID].orderedID_min; k <= d_index[gridID].orderedID_max; k++) {
                    if (orderedID == k) {
                        continue;
                    }
                    if (gpuInEpsilon_s(orderedID, k, epsilonPow, dataSize, d_dataPoints, d_ordered2DataID)) {
                        uint idx = atomicAdd(d_cnt, int(1));
                        d_orderedIDKey[idx] = orderedID;
                        d_orderedIDValue[idx] = k;
                    }
                }
            }
        }
    }
}

float Shadower::constructGPUResultSet() {
    float gpu_elapsed_time_ms;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    float *d_dataPoints;
    hipMalloc((void **)&d_dataPoints, sizeof(float) * 2 * dataSize);
    hipMemcpy(d_dataPoints, dataPoints[0].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_dataPoints + dataSize, dataPoints[1].data(), sizeof(float) * dataSize, hipMemcpyHostToDevice);

    uint *d_ordered2DataID, *d_ordered2GridID, *d_grid2CellID;
    hipMalloc((void **)&d_ordered2DataID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_ordered2GridID, sizeof(uint) * dataSize);
    hipMalloc((void **)&d_grid2CellID, sizeof(uint) * dataSize);
    hipMemcpy(d_ordered2DataID, ordered2DataID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_ordered2GridID, ordered2GridID.data(), sizeof(uint) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_grid2CellID, grid2CellID.data(), sizeof(uint) * gridSize, hipMemcpyHostToDevice);

    Grid *d_index;
    hipMalloc((void **)&d_index, sizeof(Grid) * dataSize);
    hipMemcpy(d_index, index.data(), sizeof(Grid) * dataSize, hipMemcpyHostToDevice);

    uint *d_orderedIDKey;    //key
    uint *d_orderedIDValue;  //value
    hipMalloc((void **)&d_orderedIDKey, sizeof(uint) * GPU_BUFFER_SIZE);
    hipMalloc((void **)&d_orderedIDValue, sizeof(uint) * GPU_BUFFER_SIZE);
    hipHostMalloc((void **)&orderedIDKey, sizeof(uint) * GPU_BUFFER_SIZE);
    hipHostMalloc((void **)&orderedIDValue, sizeof(uint) * GPU_BUFFER_SIZE);

    uint *d_cnt;
    hipMalloc((void **)&d_cnt, sizeof(uint));

    dim3 dimGrid(ceil((double)dataSize / (double)blockSize));
    dim3 dimBlock(blockSize);

    gpuCalcGlobal_s<<<dimGrid, dimBlock>>>(dataSize, nCells[1], gridSize, epsilonPow, d_dataPoints, d_ordered2DataID, d_ordered2GridID, d_grid2CellID, d_index, d_cnt, d_orderedIDKey, d_orderedIDValue);
    hipDeviceSynchronize();

    neighborsCnt = 0;
    hipMemcpy(&neighborsCnt, d_cnt, sizeof(uint), hipMemcpyDeviceToHost);

    // sort gpuResultSet
    device_ptr<uint> d_keyPtr(d_orderedIDKey);
    device_ptr<uint> d_valuePtr(d_orderedIDValue);

    try {
        sort_by_key(d_keyPtr, d_keyPtr + neighborsCnt, d_valuePtr);
    } catch (std::bad_alloc &e) {
        fprintf(stderr, "Error: Ran out of memory while sorting.\n");
        exit(-1);
    }

    hipMemcpy(raw_pointer_cast(orderedIDKey), raw_pointer_cast(d_keyPtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);
    hipMemcpy(raw_pointer_cast(orderedIDValue), raw_pointer_cast(d_valuePtr), sizeof(uint) * neighborsCnt, hipMemcpyDeviceToHost);

    hipFree(d_dataPoints);
    hipFree(d_ordered2DataID);
    hipFree(d_ordered2GridID);
    hipFree(d_grid2CellID);
    hipFree(d_index);
    hipFree(d_cnt);
    hipFree(d_orderedIDKey);
    hipFree(d_orderedIDValue);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, end);

    return gpu_elapsed_time_ms;
}

void Shadower::constructNeighborTable() {
    auto keyData_uqe = vector<KeyData>();
    keyData_uqe.push_back(KeyData(orderedIDKey[0], 0));
    for (uint i = 1; i < neighborsCnt; i++) {
        if (orderedIDKey[i] != orderedIDKey[i - 1]) {
            keyData_uqe.push_back(KeyData(orderedIDKey[i], i));
        }
    }

    neighborTables.resize(dataSize);
    uint key = 0;
    for (uint i = 0; i < keyData_uqe.size(); i++) {
        key = keyData_uqe[i].key;
        neighborTables[key].valueIdx_min = keyData_uqe[i].pos;
        if (i == keyData_uqe.size() - 1) {
            neighborTables[key].valueIdx_max = neighborsCnt - 1;
        } else {
            neighborTables[key].valueIdx_max = keyData_uqe[i + 1].pos - 1;
        }
    }
}

void Shadower::modifiedDBSCAN() {
    auto neighbors = vector<uint>();
    int clusterID = 0, orderedID = -1;

    for (uint i = 0; i < dataSize; i++) {
        orderedID = data2OrderedID[i];

        if (clusterIDs[i] != Shadower::UNVISITED) {
            continue;
        }
        neighbors.clear();

        uint size = neighborTables[orderedID].valueIdx_max - neighborTables[orderedID].valueIdx_min + 1;

        if ((size + 1) < minpts) {
            clusterIDs[i] = Shadower::NOISE;
            neighbors.resize(size);
            for (uint j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
                neighbors[j - neighborTables[orderedID].valueIdx_min] = orderedIDValue[j];
            }

            while (neighbors.size() != 0) {
                uint pOrderedID = neighbors.back();
                uint p = ordered2DataID[pOrderedID];
                
                int pID = clusterIDsArray[pointChunkMapping[pointIDs_shadow[p]].chunkID][pointChunkMapping[pointIDs_shadow[p]].idxInChunk];
                int iID = clusterIDsArray[pointChunkMapping[pointIDs_shadow[i]].chunkID][pointChunkMapping[pointIDs_shadow[i]].idxInChunk];
                if(merge.find(pID)==merge.end()){
                    int t = iID;
                    bool f = true;
                    while(merge.find(t)!= merge.end()){
                        if(t == pID){
                            f = false;
                            break;
                        }
                        t = merge[t];
                    }
                    if(f&&t != pID)merge.emplace(pID,t);
                }

                neighbors.pop_back();
            }
        } else {
            clusterIDs[i] = ++clusterID;
            neighbors.resize(size);
            for (uint j = neighborTables[orderedID].valueIdx_min; j <= neighborTables[orderedID].valueIdx_max; j++) {
                neighbors[j - neighborTables[orderedID].valueIdx_min] = orderedIDValue[j];
            }

            while (neighbors.size() != 0) {
                uint pOrderedID = neighbors.back();
                uint p = ordered2DataID[pOrderedID];

                if (clusterIDs[p] == Shadower::UNVISITED) {
                    uint newSize = neighborTables[pOrderedID].valueIdx_max - neighborTables[pOrderedID].valueIdx_min + 1;
                    if ((newSize + 1) >= minpts) {
                        neighbors.resize(size + newSize);
                        for (uint j = neighborTables[pOrderedID].valueIdx_min; j <= neighborTables[pOrderedID].valueIdx_max; j++) {
                            neighbors[size + j - neighborTables[pOrderedID].valueIdx_min] = orderedIDValue[j];
                        }
                    }
                }

                int pID = clusterIDsArray[pointChunkMapping[pointIDs_shadow[p]].chunkID][pointChunkMapping[pointIDs_shadow[p]].idxInChunk];
                int iID = clusterIDsArray[pointChunkMapping[pointIDs_shadow[i]].chunkID][pointChunkMapping[pointIDs_shadow[i]].idxInChunk];
                if(merge.find(pID)==merge.end()){
                    int t = iID;
                    bool f = true;
                    while(merge.find(t)!= merge.end()){
                        if(t == pID){
                            f = false;
                            break;
                        }
                        t = merge[t];
                    }
                    if(f&&t != pID)merge.emplace(pID,t);
                }
                

                if (clusterIDs[p] == Shadower::UNVISITED || clusterIDs[p] == Shadower::NOISE) {
                    clusterIDs[p] = clusterID;
                }

                neighbors.pop_back();
            }
        }
    }
}

void Shadower::run() {
    constructIndex();
    float gpu_elapsed_time_ms = constructGPUResultSet();
    constructNeighborTable();
    modifiedDBSCAN();
}